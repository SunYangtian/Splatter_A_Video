#include "hip/hip_runtime.h"
/**
 * @file compute_cov3d_kernel.cu
 * @brief CUDA kernel to compute 3D covariance matrices.
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <glm/glm.hpp>
#include <torch/torch.h>
#include <utils.h>

namespace cg = cooperative_groups;

__device__ inline glm::mat3 scale_vector_to_matrix(const glm::vec3 scale) {
    glm::mat3 S = glm::mat3(1.0f);

    S[0][0] = scale.x;
    S[1][1] = scale.y;
    S[2][2] = scale.z;

    return S;
}

__device__ inline glm::mat3
unit_quaternion_to_rotmatrix(const glm::vec4 uquat) {
    float r = uquat.x;
    float x = uquat.y;
    float y = uquat.z;
    float z = uquat.w;

    return glm::mat3(1.f - 2.f * (y * y + z * z),
                     2.f * (x * y - r * z),
                     2.f * (x * z + r * y),
                     2.f * (x * y + r * z),
                     1.f - 2.f * (x * x + z * z),
                     2.f * (y * z - r * x),
                     2.f * (x * z - r * y),
                     2.f * (y * z + r * x),
                     1.f - 2.f * (x * x + y * y));
}

__device__ void compute_cov3d_forward(const glm::vec3 scale,
                                      const glm::vec4 quat,
                                      float *cov3D) {
    glm::mat3 S = scale_vector_to_matrix(scale);
    glm::mat3 R = unit_quaternion_to_rotmatrix(quat);

    // Note: Colume Major, and right multiply.
    glm::mat3 M = S * R;
    glm::mat3 Sigma = glm::transpose(M) * M;

    cov3D[0] = Sigma[0][0];
    cov3D[1] = Sigma[0][1];
    cov3D[2] = Sigma[0][2];
    cov3D[3] = Sigma[1][1];
    cov3D[4] = Sigma[1][2];
    cov3D[5] = Sigma[2][2];
}

__device__ void compute_cov3_backward(const glm::vec3 scale,
                                      const glm::vec4 quat,
                                      const float *dL_dcov3D,
                                      glm::vec3 &dL_dscale,
                                      glm::vec4 &dL_dquat) {
    glm::mat3 S = scale_vector_to_matrix(scale);
    glm::mat3 R = unit_quaternion_to_rotmatrix(quat);
    glm::mat3 M = S * R;

    // Convert covariance loss gradients from vector to matrix
    glm::mat3 dL_dSigma = glm::mat3(dL_dcov3D[0],
                                    0.5f * dL_dcov3D[1],
                                    0.5f * dL_dcov3D[2],
                                    0.5f * dL_dcov3D[1],
                                    dL_dcov3D[3],
                                    0.5f * dL_dcov3D[4],
                                    0.5f * dL_dcov3D[2],
                                    0.5f * dL_dcov3D[4],
                                    dL_dcov3D[5]);

    // Loss gradient w.r.t. matrix M
    // dSigma_dM = 2 * M
    glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

    // Loss gradient w.r.t. scale
    glm::mat3 Rt = glm::transpose(R);
    glm::mat3 dL_dMt = glm::transpose(dL_dM);

    dL_dscale.x = glm::dot(Rt[0], dL_dMt[0]);
    dL_dscale.y = glm::dot(Rt[1], dL_dMt[1]);
    dL_dscale.z = glm::dot(Rt[2], dL_dMt[2]);

    dL_dMt[0] *= scale.x;
    dL_dMt[1] *= scale.y;
    dL_dMt[2] *= scale.z;

    // Loss gradients w.r.t. unit quaternion
    float r = quat.x;
    float x = quat.y;
    float y = quat.z;
    float z = quat.w;

    dL_dquat.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) +
                 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) +
                 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
    dL_dquat.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) +
                 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) +
                 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) -
                 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
    dL_dquat.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) +
                 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) +
                 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) -
                 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
    dL_dquat.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) +
                 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) +
                 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) -
                 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);
}

__global__ void computeCov3DForwardCUDAKernel(const int P,
                                              const glm::vec3 *scales,
                                              const glm::vec4 *uquats,
                                              const bool *visible,
                                              float *cov3Ds) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P || !visible[idx])
        return;

    compute_cov3d_forward(scales[idx], uquats[idx], cov3Ds + 6 * idx);
}

__global__ void computeCov3DBackwardCUDAKernel(const int P,
                                               const glm::vec3 *scales,
                                               const glm::vec4 *uquats,
                                               const bool *visible,
                                               const float *dL_dcov3Ds,
                                               glm::vec3 *dL_dscales,
                                               glm::vec4 *dL_duquats) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P || !visible[idx])
        return;

    compute_cov3_backward(scales[idx],
                          uquats[idx],
                          dL_dcov3Ds + 6 * idx,
                          dL_dscales[idx],
                          dL_duquats[idx]);
}

torch::Tensor computeCov3DForward(const torch::Tensor &scales,
                                  const torch::Tensor &uquats,
                                  const torch::Tensor &visible) {

    CHECK_INPUT(scales);
    CHECK_INPUT(uquats);
    CHECK_INPUT(visible);

    const int P = scales.size(0);
    auto float_opts = scales.options().dtype(torch::kFloat32);
    torch::Tensor cov3Ds = torch::zeros({P, 6}, float_opts);
    if (P != 0) {
        computeCov3DForwardCUDAKernel<<<(P + 255) / 256, 256>>>(
            P,
            (glm::vec3 *)scales.contiguous().data_ptr<float>(),
            (glm::vec4 *)uquats.contiguous().data_ptr<float>(),
            visible.contiguous().data_ptr<bool>(),
            cov3Ds.data_ptr<float>());
    }

    return cov3Ds;
}

std::tuple<torch::Tensor, torch::Tensor>
computeCov3DBackward(const torch::Tensor &scales,
                     const torch::Tensor &uquats,
                     const torch::Tensor &visible,
                     const torch::Tensor &dL_dcov3Ds) {

    CHECK_INPUT(scales);
    CHECK_INPUT(uquats);
    CHECK_INPUT(visible);
    CHECK_INPUT(dL_dcov3Ds);

    const int P = scales.size(0);
    auto float_opts = scales.options().dtype(torch::kFloat32);
    torch::Tensor dL_dscales = torch::zeros({P, 3}, float_opts);
    torch::Tensor dL_duquats = torch::zeros({P, 4}, float_opts);

    if (P != 0) {
        computeCov3DBackwardCUDAKernel<<<(P + 255) / 256, 256>>>(
            P,
            (glm::vec3 *)scales.contiguous().data_ptr<float>(),
            (glm::vec4 *)uquats.contiguous().data_ptr<float>(),
            visible.contiguous().data_ptr<bool>(),
            dL_dcov3Ds.contiguous().data_ptr<float>(),
            (glm::vec3 *)dL_dscales.data_ptr<float>(),
            (glm::vec4 *)dL_duquats.data_ptr<float>());
    }

    return std::make_tuple(dL_dscales, dL_duquats);
}