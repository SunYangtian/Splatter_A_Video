#include "hip/hip_runtime.h"
/**
 * @file compute_sh.cu
 * @brief
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <glm/glm.hpp>
#include <torch/torch.h>
#include <utils.h>


namespace cg = cooperative_groups;

__device__ const float SH_C0 = 0.28209479177387814f;
__device__ const float SH_C1 = 0.4886025119029199f;
__device__ const float SH_C2[] = {1.0925484305920792f,
                                  -1.0925484305920792f,
                                  0.31539156525252005f,
                                  -1.0925484305920792f,
                                  0.5462742152960396f};
__device__ const float SH_C3[] = {-0.5900435899266435f,
                                  2.890611442640554f,
                                  -0.4570457994644658f,
                                  0.3731763325901154f,
                                  -0.4570457994644658f,
                                  1.445305721320277f,
                                  -0.5900435899266435f};

__device__ const unsigned num_sh_bases[] = {1, 4, 9, 16};

__global__ void computeSHForwardCUDAKernel(const int P,
                                           const glm::vec3 *shs,
                                           const int deg,
                                           const glm::vec3 *dirs,
                                           const bool *visible,
                                           glm::vec3 *colors,
                                           bool *clamped) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P || !visible[idx])
        return;

    glm::vec3 dir = dirs[idx];

    const glm::vec3 *sh = shs + idx * num_sh_bases[deg];
    glm::vec3 result = SH_C0 * sh[0];

    if (deg > 0) {
        float x = dir.x;
        float y = dir.y;
        float z = dir.z;
        result =
            result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

        if (deg > 1) {
            float xx = x * x, yy = y * y, zz = z * z;
            float xy = x * y, yz = y * z, xz = x * z;
            result = result + SH_C2[0] * xy * sh[4] + SH_C2[1] * yz * sh[5] +
                     SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
                     SH_C2[3] * xz * sh[7] + SH_C2[4] * (xx - yy) * sh[8];

            if (deg > 2) {
                result = result + SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
                         SH_C3[1] * xy * z * sh[10] +
                         SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
                         SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) *
                             sh[12] +
                         SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
                         SH_C3[5] * z * (xx - yy) * sh[14] +
                         SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
            }
        }
    }
    result += 0.5f;

    clamped[3 * idx + 0] = (result.x < 0);
    clamped[3 * idx + 1] = (result.y < 0);
    clamped[3 * idx + 2] = (result.z < 0);
    colors[idx] = glm::max(result, 0.0f);
}

__global__ void computeSHBackwardCUDAKernel(const int P,
                                            const glm::vec3 *shs,
                                            const int deg,
                                            const glm::vec3 *dirs,
                                            const bool *visible,
                                            const bool *clamped,
                                            const glm::vec3 *dL_dcolors,
                                            glm::vec3 *dL_dshs,
                                            glm::vec3 *dL_ddirs) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P || !visible[idx])
        return;

    glm::vec3 dir = dirs[idx];

    const glm::vec3 *sh = shs + idx * num_sh_bases[deg];

    glm::vec3 dL_dcolor = dL_dcolors[idx];
    dL_dcolor.x *= clamped[3 * idx + 0] ? 0 : 1;
    dL_dcolor.y *= clamped[3 * idx + 1] ? 0 : 1;
    dL_dcolor.z *= clamped[3 * idx + 2] ? 0 : 1;

    glm::vec3 dRGBdx(0, 0, 0);
    glm::vec3 dRGBdy(0, 0, 0);
    glm::vec3 dRGBdz(0, 0, 0);
    float x = dir.x;
    float y = dir.y;
    float z = dir.z;

    glm::vec3 *dL_dsh = dL_dshs + idx * num_sh_bases[deg];

    float dRGBdsh0 = SH_C0;
    dL_dsh[0] = dRGBdsh0 * dL_dcolor;
    if (deg > 0) {
        float dRGBdsh1 = -SH_C1 * y;
        float dRGBdsh2 = SH_C1 * z;
        float dRGBdsh3 = -SH_C1 * x;
        dL_dsh[1] = dRGBdsh1 * dL_dcolor;
        dL_dsh[2] = dRGBdsh2 * dL_dcolor;
        dL_dsh[3] = dRGBdsh3 * dL_dcolor;

        dRGBdx = -SH_C1 * sh[3];
        dRGBdy = -SH_C1 * sh[1];
        dRGBdz = SH_C1 * sh[2];

        if (deg > 1) {
            float xx = x * x, yy = y * y, zz = z * z;
            float xy = x * y, yz = y * z, xz = x * z;

            float dRGBdsh4 = SH_C2[0] * xy;
            float dRGBdsh5 = SH_C2[1] * yz;
            float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
            float dRGBdsh7 = SH_C2[3] * xz;
            float dRGBdsh8 = SH_C2[4] * (xx - yy);
            dL_dsh[4] = dRGBdsh4 * dL_dcolor;
            dL_dsh[5] = dRGBdsh5 * dL_dcolor;
            dL_dsh[6] = dRGBdsh6 * dL_dcolor;
            dL_dsh[7] = dRGBdsh7 * dL_dcolor;
            dL_dsh[8] = dRGBdsh8 * dL_dcolor;

            dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] +
                      SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
            dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] +
                      SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
            dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] +
                      SH_C2[3] * x * sh[7];

            if (deg > 2) {
                float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
                float dRGBdsh10 = SH_C3[1] * xy * z;
                float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
                float dRGBdsh12 =
                    SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
                float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
                float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
                float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
                dL_dsh[9] = dRGBdsh9 * dL_dcolor;
                dL_dsh[10] = dRGBdsh10 * dL_dcolor;
                dL_dsh[11] = dRGBdsh11 * dL_dcolor;
                dL_dsh[12] = dRGBdsh12 * dL_dcolor;
                dL_dsh[13] = dRGBdsh13 * dL_dcolor;
                dL_dsh[14] = dRGBdsh14 * dL_dcolor;
                dL_dsh[15] = dRGBdsh15 * dL_dcolor;

                dRGBdx +=
                    (SH_C3[0] * sh[9] * 3.f * 2.f * xy +
                     SH_C3[1] * sh[10] * yz + SH_C3[2] * sh[11] * -2.f * xy +
                     SH_C3[3] * sh[12] * -3.f * 2.f * xz +
                     SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
                     SH_C3[5] * sh[14] * 2.f * xz +
                     SH_C3[6] * sh[15] * 3.f * (xx - yy));

                dRGBdy += (SH_C3[0] * sh[9] * 3.f * (xx - yy) +
                           SH_C3[1] * sh[10] * xz +
                           SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
                           SH_C3[3] * sh[12] * -3.f * 2.f * yz +
                           SH_C3[4] * sh[13] * -2.f * xy +
                           SH_C3[5] * sh[14] * -2.f * yz +
                           SH_C3[6] * sh[15] * -3.f * 2.f * xy);

                dRGBdz += (SH_C3[1] * sh[10] * xy +
                           SH_C3[2] * sh[11] * 4.f * 2.f * yz +
                           SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
                           SH_C3[4] * sh[13] * 4.f * 2.f * xz +
                           SH_C3[5] * sh[14] * (xx - yy));
            }
        }
    }

    glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dcolor),
                      glm::dot(dRGBdy, dL_dcolor),
                      glm::dot(dRGBdz, dL_dcolor));
    dL_ddirs[idx] = dL_ddir;
}

void computeSHForwardCUDA(const int P,
                          const float *shs,
                          const int deg,
                          const float *dirs,
                          const bool *visible,
                          float *colors,
                          bool *clamped) {
    computeSHForwardCUDAKernel<<<(P + 255) / 256, 256>>>(
        P,
        (const glm::vec3 *)shs,
        deg,
        (const glm::vec3 *)dirs,
        visible,
        (glm::vec3 *)colors,
        clamped);
}

void computeSHBackwardCUDA(const int P,
                           const float *shs,
                           const int deg,
                           const float *dirs,
                           const bool *visible,
                           const bool *clamped,
                           const float *dL_dcolors,
                           float *dL_dshs,
                           float *dL_ddirs) {
    computeSHBackwardCUDAKernel<<<(P + 255) / 256, 256>>>(
        P,
        (const glm::vec3 *)shs,
        deg,
        (const glm::vec3 *)dirs,
        visible,
        clamped,
        (glm::vec3 *)dL_dcolors,
        (glm::vec3 *)dL_dshs,
        (glm::vec3 *)dL_ddirs);
}

std::tuple<torch::Tensor, torch::Tensor>
computeSHForward(const torch::Tensor &shs,
                 const int degree,
                 const torch::Tensor &view_dirs,
                 const torch::Tensor &visible) {
    CHECK_INPUT(shs);
    CHECK_INPUT(view_dirs);
    CHECK_INPUT(visible);

    const int P = shs.size(0);
    auto float_opts = shs.options().dtype(torch::kFloat32);
    auto bool_ops = shs.options().dtype(torch::kBool);
    torch::Tensor colors = torch::zeros({P, 3}, float_opts);
    torch::Tensor clamped = torch::ones({P, 3}, bool_ops);

    if (P != 0) {
        computeSHForwardCUDAKernel<<<(P + 255) / 256, 256>>>(
            P,
            (const glm::vec3 *)shs.contiguous().data_ptr<float>(),
            degree,
            (const glm::vec3 *)view_dirs.contiguous().data_ptr<float>(),
            visible.contiguous().data_ptr<bool>(),
            (glm::vec3 *)colors.data_ptr<float>(),
            clamped.data_ptr<bool>());
    }

    return std::make_tuple(colors, clamped);
}

std::tuple<torch::Tensor, torch::Tensor>
computeSHBackward(const torch::Tensor &shs,
                  const int degree,
                  const torch::Tensor &view_dirs,
                  const torch::Tensor &visible,
                  const torch::Tensor &clamped,
                  const torch::Tensor &dL_dcolors) {
    CHECK_INPUT(shs);
    CHECK_INPUT(view_dirs);
    CHECK_INPUT(visible);
    CHECK_INPUT(dL_dcolors);

    const int P = shs.size(0);
    const int S = shs.size(1);
    auto float_opts = shs.options().dtype(torch::kFloat32);
    torch::Tensor dL_dshs = torch::zeros({P, S, 3}, float_opts);
    torch::Tensor dL_dvdirs = torch::zeros({P, 3}, float_opts);

    if (P != 0) {
        computeSHBackwardCUDAKernel<<<(P + 255) / 256, 256>>>(
            P,
            (const glm::vec3 *)shs.contiguous().data_ptr<float>(),
            degree,
            (const glm::vec3 *)view_dirs.contiguous().data_ptr<float>(),
            visible.contiguous().data_ptr<bool>(),
            clamped.contiguous().data_ptr<bool>(),
            (glm::vec3 *)dL_dcolors.contiguous().data_ptr<float>(),
            (glm::vec3 *)dL_dshs.data_ptr<float>(),
            (glm::vec3 *)dL_dvdirs.data_ptr<float>());
    }

    return std::make_tuple(dL_dshs, dL_dvdirs);
}